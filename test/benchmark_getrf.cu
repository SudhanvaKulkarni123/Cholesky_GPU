#include <iostream>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#define M 4096  // Number of rows (tall)
#define N 512    // Number of columns (skinny)
#define NUM_RUNS 10  // Number of iterations for averaging performance

#define CUDA_CHECK(call)                                                    \
    {                                                                       \
        hipError_t err = call;                                             \
        if (err != hipSuccess) {                                           \
            std::cerr << "CUDA Error: " << hipGetErrorString(err) << " at " \
                      << __FILE__ << ":" << __LINE__ << std::endl;          \
            exit(EXIT_FAILURE);                                             \
        }                                                                   \
    }

#define CUSOLVER_CHECK(call)                                                \
    {                                                                       \
        hipsolverStatus_t err = call;                                        \
        if (err != HIPSOLVER_STATUS_SUCCESS) {                               \
            std::cerr << "cuSOLVER Error at " << __FILE__ << ":" << __LINE__ \
                      << std::endl;                                         \
            exit(EXIT_FAILURE);                                             \
        }                                                                   \
    }

int main() {
    hipsolverHandle_t cusolverH;
    (hipsolverDnCreate(&cusolverH));

    float *d_A;
    int *d_pivot, *d_info;
    int lwork = 0;
    float *d_work;

    // Allocate matrix (row-major layout: MxN)
    CUDA_CHECK(hipMalloc((void**)&d_A, M * N * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&d_pivot, N * sizeof(int)));  // Only N pivots
    CUDA_CHECK(hipMalloc((void**)&d_info, sizeof(int)));       // For error check

    // Query workspace size for GETRF
    CUSOLVER_CHECK(hipsolverDnSgetrf_bufferSize(cusolverH, M, N, d_A, M, &lwork));

    // Allocate workspace
    CUDA_CHECK(hipMalloc((void**)&d_work, lwork * sizeof(float)));

    // Create CUDA events for timing
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // Benchmark cuSOLVER GETRF over multiple runs
    float total_time_ms = 0.0f;
    for (int i = 0; i < NUM_RUNS; i++) {
        CUDA_CHECK(hipEventRecord(start));

        // Perform LU factorization (in-place)
        CUSOLVER_CHECK(hipsolverDnSgetrf(cusolverH, M, N, d_A, M, d_work, d_pivot, d_info));

        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));

        float elapsed_ms;
        CUDA_CHECK(hipEventElapsedTime(&elapsed_ms, start, stop));
        total_time_ms += elapsed_ms;
    }

    // Compute average time and GFLOPS
    float avg_time_ms = total_time_ms / NUM_RUNS;
    double gflops = (2.0 * M * N * N) / (3.0 * avg_time_ms * 1e6); // Approximate GFLOPS formula

    std::cout << "cuSOLVER GETRF Performance for " << M << "x" << N << " matrix:\n";
    std::cout << "  Avg Time: " << avg_time_ms << " ms\n";
    std::cout << "  GFLOPS: " << gflops << " GFLOP/s\n";

    // Cleanup
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_pivot));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_work));
 (hipsolverDnDestroy(cusolverH));
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    return 0;
}
