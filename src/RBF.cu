#include "hip/hip_runtime.h"
///@author Sudhanva Kulkarni
/// this file contains code to generat the diagonal RBF kernel and being cholesky
#include <hip/hip_runtime.h>
#include <stdio>

#define CUDA_CHECK(err)         \
    do {                        \
        hipError_t e = (err);   \
        if(e != hipSuccess) {  \
        fprintf(stderr, "CUDA error %s, at %d : %s",            \
        hipGetErrorString(e), __LINE__, __FILE__);\
            exit(EXIT_FAILURE);}         \
    } while(0)                  \


__global__ void double_array(float* __restrict__ A, int n) {
    for(int i = threadIdx.x + blockIdx.x * blockDim.x; i < n; i += gridDim.x * blockDim.x)
    {
        A[i] *= 2.0f;
    }
}



int main() {

    int n = 50;
    float* h_arr = nullptr;
    hipHostAlloc((void**) &h_arr, n*sizeof(float), hipHostMallocDefault);

    float* d_arr = nullptr;
    hipMalloc((void**)&d_arr, n*sizeof(float));

    for(int i = 0; i < n; i++) h_arr[i] = 1.0f;

    hipMemcpy(d_arr, h_arr, n*sizeof(float), hipMemcpyHostToDevice);

    int BlockSize = 1024;
    int GridSize = n/1024;

    hipStream_t stream;
    

}

