#include "hip/hip_runtime.h"
#include "kernels_macros.cuh"
#include "matgen.cuh"
#include "micro_chol.hpp"
#include <fstream>
#include <iostream>
#include <cutlass/epilogue/thread/linear_combination_clamp.h>
#include <json.hpp>
//#include "sChol.cuh"
//#include "hChol.cuh"
//#include "fp8Chol.cuh"
#include "switching_chol.cuh"
#include <mkl.h>


using namespace std;

enum distr_type : uint8_t {
    Arith, Geom, Rand
};

void host_isnan(float* A, int n)
{
    for(int i = 0; i < n*n; i++) {
        if(isnanf(A[i])){ std::cout << "A has a nan\n"; return;}

    }
}


// vanilla CG
int vanilla_CG(double* A, double* x, double* b, int n, double normA)
{

}


// Preconditioned Conjugate Gradient (CG) solver (simplified skeleton)
// This function demonstrates the allocation, conversion, and Cholesky preconditioning.
// The actual CG loop here is simplified and uses dummy updates.
//-----------------------------
// precond_CG function
//-----------------------------
int precond_CG(double* A, double* x,
               double* b, int n, int r, double normA, float eps_prime = 0.0f, float flr = 0.0f) {


    
    std::ofstream myfile("e5m2_error_f_cond.csv");
    hipEvent_t start, stop;
        CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));


    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));  // Still creating a stream for kernels/BLAS if we like
    hipStream_t logging_stream;
    CUDA_CHECK(hipStreamCreate(&logging_stream));
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));
    CUBLAS_CHECK(hipblasSetStream(handle, stream));

    hipsolverHandle_t CuHandle_t;
    CUSOLVER_CHECK(hipsolverDnCreate(&CuHandle_t));
    CUSOLVER_CHECK(hipsolverSetStream(CuHandle_t, stream));

    


    double One = 1.0;
    float Onef = 1.0f;
    double Zero = 0.0;
    double NegOne = -1.0;

    
    hipEvent_t init_start, init_stop;
     CUDA_CHECK(hipEventCreate(&init_start));
    CUDA_CHECK(hipEventCreate(&init_stop));
    CUDA_CHECK(hipEventRecord(init_start, stream)); 
// Compute total memory required
size_t size_A_d = n * n * sizeof(double);
size_t size_A_f = n * n * sizeof(float);
size_t max_buff_size = n * r * sizeof(float);  // Matul buffer size
size_t vec_size = n * sizeof(double);
size_t half_vec_size = n * sizeof(float);  // For float vectors
size_t scal_size = sizeof(double);
size_t perm_size = n * sizeof(int);



// Total size to allocate in a single call
size_t workspace_size = n * n * sizeof(float);  // New workspace buffer

size_t total_size = 
    size_A_d +  // d_A
    size_A_f +  // s_A
    workspace_size + // New workspace buffer
    size_A_d +  // LL_copy
    7 * vec_size +  // dev_x, dev_b, dev_r, p, Ap, dev_z, dev_D (all double)
    2 * half_vec_size +  // diag_A, updated_diag (float)
    4 * scal_size +  // pAp, rz, rz_new, inf_norm_r (double)
    3 * scal_size +  // d_One, d_Zero, d_NegOne (double)
    2 * perm_size + sizeof(int);  // dev_left_perm, dev_right_perm (int)

// Allocate memory
void* d_mem = nullptr;
CUDA_CHECK(hipMalloc(&d_mem, total_size));

char* d_mem_char = reinterpret_cast<char*>(d_mem);

double* d_A        = reinterpret_cast<double*>(d_mem_char);
double* LL_copy    = reinterpret_cast<double*>(d_mem_char + size_A_d);  // LL_copy right after d_A
float* s_A         = reinterpret_cast<float*>(d_mem_char + size_A_d + size_A_d);
float* workspace   = reinterpret_cast<float*>(d_mem_char + size_A_d + size_A_d + size_A_f);  // Workspace right after s_A

double* dev_x      = reinterpret_cast<double*>(d_mem_char + size_A_d + size_A_d + size_A_f + workspace_size);
double* dev_b      = dev_x + n;
double* dev_r      = dev_b + n;
double* p          = dev_r + n;
double* Ap         = p + n;
double* dev_z      = Ap + n;
double* dev_D      = dev_z + n;

float* diag_A      = reinterpret_cast<float*>(dev_D + n);  
float* updated_diag= diag_A + n;

double* pAp        = reinterpret_cast<double*>(updated_diag + n);
double* rz         = pAp + 1;
double* rz_new     = rz + 1;
double* inf_norm_r = rz_new + 1;

double* d_One      = inf_norm_r + 1;
double* d_Zero     = d_One + 1;
double* d_NegOne   = d_Zero + 1;

int* dev_left_perm  = reinterpret_cast<int*>(d_NegOne + 1);
int* dev_right_perm = dev_left_perm + n;
int* devInfo = dev_right_perm + n;


// Copy scalar values asynchronously
CUDA_CHECK(hipMemcpyAsync(d_One,    &One,    scal_size, hipMemcpyHostToDevice, stream));
CUDA_CHECK(hipMemcpyAsync(d_Zero,   &Zero,   scal_size, hipMemcpyHostToDevice, stream));
CUDA_CHECK(hipMemcpyAsync(d_NegOne, &NegOne, scal_size, hipMemcpyHostToDevice, stream));

// Allocate host memory for preconditioner
double* D_host       = (double*)malloc(n * sizeof(double));
int* left_perm_host  = (int*)malloc(n * sizeof(int));
int* right_perm_host = (int*)malloc(n * sizeof(int));

// Initialize preconditioner data
for (int i = 0; i < n; i++) {
    left_perm_host[i] = i;
    right_perm_host[i] = i;
    D_host[i] = 1.0;  // Example
}

// Copy permutation and preconditioner data asynchronously
CUDA_CHECK(hipMemcpyAsync(dev_left_perm, left_perm_host, perm_size, hipMemcpyHostToDevice, stream));
CUDA_CHECK(hipMemcpyAsync(dev_right_perm, right_perm_host, perm_size, hipMemcpyHostToDevice, stream));
CUDA_CHECK(hipMemcpyAsync(dev_D, D_host, vec_size, hipMemcpyHostToDevice, stream));

    // Copy matrix A and vector b from host to device (synchronous).
    CUDA_CHECK(hipMemcpyAsync(d_A, A, size_A_d, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));

    CUDA_CHECK(hipMemcpyAsync(dev_b, b, vec_size, hipMemcpyHostToDevice, stream));

    // Initialize search direction p with b (synchronous).
    CUDA_CHECK(hipMemcpyAsync(p, b, vec_size, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));

    // Convert double-precision matrix to single precision (kernel uses stream).
    int totalElems = n * n;
    int threads = 256;
    int blocks = (totalElems + threads - 1) / threads;
    convertDoubleToFloat<<<blocks, threads, 0, stream>>>(d_A, s_A, totalElems);
    CUDA_CHECK(hipStreamSynchronize(stream));


    blocks = (n + threads - 1)/threads;
    copy_diag<<<blocks, threads, 0, stream>>>(diag_A, updated_diag, s_A, n);
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipEventRecord(init_stop, stream));
    CUDA_CHECK(hipEventSynchronize(init_stop));
    float time_init = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&time_init, init_start, init_stop));
    printf("initializartion took %.2f ms.\n", time_init);
    // hipEventRecord(stop);
    // CUDA_CHECK(hipEventSynchronize(stop));
    // float time_init = 0.0f;
    // CUDA_CHECK(hipEventElapsedTime(&time_init, start, stop));
    // printf("initialization took %.2f ms.\n", time_init);



    // Perform mixed-precision Cholesky factorization on s_A (with your function).



    CUDA_CHECK(hipEventRecord(start, stream));
    CUDA_CHECK(hipDeviceSynchronize());
    //uniform_prec_GPU_cholesky(s_A, n, r, diag_A, updated_diag, handle, CuHandle_t ,stream, n);
    //uniform_prec_fused_cholesky(s_A, n, r, diag_A, updated_diag, handle, CuHandle_t ,stream, n);
    //uniform_prec_GPU_cholesky(s_A, n, r, diag_A, updated_diag, handle, CuHandle_t ,stream, n);
    //uniform_precision_Cholesky(s_A, n, r, diag_A, updated_diag, handle, stream, n);
   // halfprec_mixed_precision_Cholesky(s_A, n, r, diag_A, updated_diag, handle, CuHandle_t, stream, n, reinterpret_cast<cutlass::half_t*>(workspace));
    //fp8_mixed_precision_Cholesky(s_A, n, r, diag_A, updated_diag, handle, CuHandle_t, stream, n, reinterpret_cast<cutlass::float_e4m3_t*>(workspace), flr);

    switching_precision_Cholesky(s_A, n, r, workspace, diag_A, updated_diag, handle, CuHandle_t, stream, n, eps_prime, flr);
    CUDA_CHECK(hipStreamSynchronize(stream));
      CUDA_CHECK(hipEventRecord(stop, stream));
    CUDA_CHECK(hipEventSynchronize(stop));
    float time_our = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&time_our, start, stop));
    printf("mixed precision cholesky took %.2f ms.\n", time_our);

    //perform trmm to see if we get back same matrix

    #ifdef DEBUG_MODE
    float* debug_mat = nullptr;
    float* host_debug_mat = (float*) malloc(n*n*sizeof(float));
    CUDA_CHECK(hipMalloc((void**) &debug_mat, n*n*sizeof(float)));
    CUDA_CHECK(hipMemset(debug_mat, 0.0f, n*n*sizeof(float)));
    set_identity<<<blocks, threads, 0, stream>>>(debug_mat, n);
    CUBLAS_CHECK(hipblasStrmm(
        handle,
        HIPBLAS_SIDE_LEFT,
        HIPBLAS_FILL_MODE_LOWER,
        HIPBLAS_OP_N,
        HIPBLAS_DIAG_NON_UNIT,
        n,          // # of rows of debug_mat
        n,          // # of columns of debug_mat
        &Onef,
        s_A, n,     // pointer to L
        debug_mat, n,
        debug_mat, n
    ));

    CUDA_CHECK(hipStreamSynchronize(stream));
    CUBLAS_CHECK(hipblasStrmm(
        handle,
        HIPBLAS_SIDE_LEFT,
        HIPBLAS_FILL_MODE_LOWER,
        HIPBLAS_OP_T,      // now use the transpose of L
        HIPBLAS_DIAG_NON_UNIT,
        n,  // # rows
        n,  // # cols
        &Onef,
        s_A, n,
        debug_mat, n,
        debug_mat, n
    ));
    CUDA_CHECK(hipStreamSynchronize(stream));



    #endif

    CUDA_CHECK(hipStreamSynchronize(stream));
    

    CUBLAS_CHECK(hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE));


    // Convert factorized single-precision matrix back to double
    convertFloattoDouble<<<blocks, threads, 0, stream>>>(s_A, LL_copy, totalElems);
    CUDA_CHECK(hipStreamSynchronize(stream));

    // Solve L y = b then L^T x = y; use p as temp
    float time_init_soln;
    hipEventRecord(start, stream);
    CUSOLVER_CHECK(hipsolverDnDpotrs(CuHandle_t, HIPBLAS_FILL_MODE_LOWER, n, 1, LL_copy, n, dev_z, n, devInfo));

    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_init_soln, start, stop);


    // Set dev_x = p (device to device copy - synchronous).
    CUDA_CHECK(hipMemcpy(dev_x, p, vec_size, hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipStreamSynchronize(stream));

    // r = b - A*x
    CUBLAS_CHECK(hipblasDgemv(handle, HIPBLAS_OP_N, n, n, d_NegOne,
                             d_A, n, dev_x, 1, d_One, dev_b, 1));           //doesnt matter if its row or col major
    CUDA_CHECK(hipMemcpy(dev_r, dev_b, vec_size, hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipStreamSynchronize(stream));

     double cpu_inf_norm_r = 0.0;

    int vecThreads = 256;
    int vecBlocks = (n + vecThreads - 1) / vecThreads;

        // Allocate temporary device memory to store per-block maximums.
    double* dev_blockMax = nullptr;
    CUDA_CHECK(hipMalloc((void**)&dev_blockMax, vecBlocks * sizeof(double)));

    vanilla_Max<<<vecBlocks, vecThreads, vecThreads * sizeof(double), stream>>>(dev_r, dev_blockMax, n);

    int finalThreads = 256;

    final_max_reduce<<<1, finalThreads, finalThreads * sizeof(double), stream>>>(dev_blockMax, inf_norm_r, vecBlocks);

    // Wait for the kernels to finish.
    CUDA_CHECK(hipStreamSynchronize(stream));

    // Free the temporary block maximum array.
    CUDA_CHECK(hipFree(dev_blockMax));

    CUDA_CHECK(hipStreamSynchronize(stream));

       
    // synchronous memcpy from device to host
    CUDA_CHECK(hipMemcpy(&cpu_inf_norm_r, inf_norm_r, sizeof(double), hipMemcpyDeviceToHost));

    double init_residual = cpu_inf_norm_r / normA;
    std::cout << "init residual is : " << init_residual << std::endl;


    float time_init_precond;
    // Compute r^T z
    hipEventRecord(start, stream);
    CUDA_CHECK(hipMemcpy(dev_z, dev_r, vec_size, hipMemcpyDeviceToDevice));
    CUSOLVER_CHECK(hipsolverDnDpotrs(CuHandle_t, HIPBLAS_FILL_MODE_LOWER, n, 1, LL_copy, n, dev_z, n, devInfo));


    CUBLAS_CHECK(hipblasDdot(handle, n, dev_z, 1, dev_r, 1, rz));

    CUDA_CHECK(hipMemcpy(p, dev_z, vec_size, hipMemcpyDeviceToDevice));
    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_init_precond, start, stop);

    const double d_conv_bound = 1e-14;
    int count = 0;

    float gemv_CG_time = 0;
    float trsv_CG_time = 0;

    float update_search_time = 0;

    // CG loop
    const int max_CG_iter = 1000;
    for (int j = 0; j < max_CG_iter; j++) {
        // Ap = A * p
        hipEventRecord(start, stream);
        CUBLAS_CHECK(hipblasDgemv(handle, HIPBLAS_OP_N, n, n,
                                 d_One, d_A, n, p, 1, d_Zero, Ap, 1));          //doesnt matter if its row or col major
        
        CUBLAS_CHECK(hipblasDdot(handle, n, Ap, 1, p, 1, pAp));
        CUDA_CHECK(hipStreamSynchronize(stream));
        hipEventRecord(stop, stream);
        hipEventSynchronize(stop);
        float elapsed_time;
        hipEventElapsedTime(&elapsed_time, start, stop);
        gemv_CG_time += elapsed_time;



        // x_r_update kernel
  
        x_r_update<<<vecBlocks, vecThreads, sizeof(double), stream>>>(dev_x, dev_r, rz, pAp, p, Ap, n);
        CUDA_CHECK(hipStreamSynchronize(stream));

        // Compute infinity norm of residual
         vanilla_Max<<<vecBlocks, vecThreads, vecThreads * sizeof(double), stream>>>(dev_r, dev_blockMax, n);
         final_max_reduce<<<1, finalThreads, finalThreads * sizeof(double), stream>>>(dev_blockMax, inf_norm_r, vecBlocks);
        CUDA_CHECK(hipStreamSynchronize(stream));

       
        // synchronous memcpy from device to host
        CUDA_CHECK(hipMemcpy(&cpu_inf_norm_r, inf_norm_r, sizeof(double), hipMemcpyDeviceToHost));

        double scaled_residual = cpu_inf_norm_r / normA;
        myfile << j << "," << scaled_residual << std::endl;
        if (scaled_residual < d_conv_bound) {
            count = j + 1;
            std::cout << "Converged in " << count << " iterations." << std::endl;

            break;
        }

        // Preconditioning
 
        CUDA_CHECK(hipMemcpy(dev_z, dev_r, vec_size, hipMemcpyDeviceToDevice));
        CUDA_CHECK(hipStreamSynchronize(stream));

        // Scale dev_z by D
        //diag_scal<<<vecBlocks, vecThreads, 0, stream>>>(dev_z, dev_z, dev_D, n);
        CUDA_CHECK(hipStreamSynchronize(stream));

        hipEventRecord(start, stream);
        CUSOLVER_CHECK(hipsolverDnDpotrs(CuHandle_t, HIPBLAS_FILL_MODE_LOWER, n, 1, LL_copy, n, dev_z, n, devInfo));
        hipEventRecord(stop, stream);
        hipEventSynchronize(stop);
        float elapsed_time2;
        hipEventElapsedTime(&elapsed_time2, start, stop);
        trsv_CG_time += elapsed_time2;

        // Scale dev_z by D again
        // diag_scal<<<vecBlocks, vecThreads, 0, stream>>>(dev_z, dev_z, dev_D, n);
        CUDA_CHECK(hipStreamSynchronize(stream));

        // Compute r^T z
        CUBLAS_CHECK(hipblasDdot(handle, n, dev_r, 1, dev_z, 1, rz_new));

        // p = z + beta * p
        hipEventRecord(start, stream);
        update_search_dir<<<vecBlocks, vecThreads, sizeof(double), stream>>>(p, dev_z, rz_new, rz, n);
        CUDA_CHECK(hipStreamSynchronize(stream));
        hipEventRecord(stop, stream);
        hipEventSynchronize(stop);
        float elapsed_time3;
        hipEventElapsedTime(&elapsed_time3, start, stop);
        update_search_time += elapsed_time3;

        CUDA_CHECK(hipMemcpy(rz, rz_new, sizeof(double), hipMemcpyDeviceToDevice));
        CUDA_CHECK(hipStreamSynchronize(stream));
        count++;
    }


    printf("GEMVs in CG took %.2f ms.\n", gemv_CG_time);
    printf("trsv in CG took %.2f ms.\n", trsv_CG_time);
    printf("search dir update in CG took %.2f ms.\n", update_search_time);




    float epilogue_time;
    // Copy final solution to host
    CUDA_CHECK(hipMemcpy(x, dev_x, vec_size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipDeviceSynchronize());

    // Clean up
   CUDA_CHECK(hipFree(d_mem));

    CUBLAS_CHECK(hipblasDestroy(handle));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipStreamDestroy(logging_stream));



    myfile.close();

    return count;
    } // End CG loop.

// Solve A*x = b using cuSOLVER's Cholesky factorization.
int solveWithCuSolver(double* A, double* x, const double* b, int n) {
    hipsolverHandle_t cusolverH = nullptr;
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
    
    double *d_A = nullptr;
    double *d_b = nullptr;
    int lda = n;
    size_t matrixSize = n * n * sizeof(double);
    size_t vecSize = n * sizeof(double);
    CUDA_CHECK(hipMalloc((void**)&d_A, matrixSize));
    CUDA_CHECK(hipMalloc((void**)&d_b, vecSize));
    
    // Copy A and b from host to device.
    CUDA_CHECK(hipMemcpy(d_A, A, matrixSize, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, b, vecSize, hipMemcpyHostToDevice));
    
    int work_size = 0;
    int *devInfo = nullptr;
    CUDA_CHECK(hipMalloc((void**)&devInfo, sizeof(int)));
    
    // Query working space for Cholesky factorization.
    hipsolverDnDpotrf_bufferSize(cusolverH, HIPBLAS_FILL_MODE_LOWER, n, d_A, lda, &work_size);
    double *work = nullptr;
    CUDA_CHECK(hipMalloc((void**)&work, work_size * sizeof(double)));
    
    // Cholesky factorization: A = L * L^T.
    hipsolverDnDpotrf(cusolverH, HIPBLAS_FILL_MODE_LOWER, n, d_A, lda, work, work_size, devInfo);
    int devInfo_h = 0;
    CUDA_CHECK(hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    if (devInfo_h != 0) {
        printf("Cholesky factorization failed with devInfo = %d\n", devInfo_h);
        exit(EXIT_FAILURE);
    }
    
    // Solve A*x = b.
    hipsolverDnDpotrs(cusolverH, HIPBLAS_FILL_MODE_LOWER, n, 1, d_A, lda, d_b, n, devInfo);
    CUDA_CHECK(hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    if (devInfo_h != 0) {
        printf("Cholesky solve failed with devInfo = %d\n", devInfo_h);
        exit(EXIT_FAILURE);
    }
    
    // Copy the solution from device to host.
    CUDA_CHECK(hipMemcpy(x, d_b, vecSize, hipMemcpyDeviceToHost));
    
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(work));
    CUDA_CHECK(hipFree(devInfo));
    hipsolverDnDestroy(cusolverH);
    
    return 0;
}


// Solve A*x = b using Intel MKL's Cholesky factorization.
int solveWithMKL(double* A, double* x, const double* b, int n) {
    // Copy b to x since LAPACK overwrites b with the solution.
    memcpy(x, b, n * sizeof(double));

    // Perform Cholesky factorization (A = L * L^T)
    int info = LAPACKE_dpotrf(LAPACK_COL_MAJOR, 'L', n, A, n);
    if (info != 0) {
        printf("MKL Cholesky factorization failed with info = %d\n", info);
        exit(EXIT_FAILURE);
    }

    // Solve A*x = b using the factorized matrix
    info = LAPACKE_dpotrs(LAPACK_COL_MAJOR, 'L', n, 1, A, n, x, n);
    if (info != 0) {
        printf("MKL Cholesky solve failed with info = %d\n", info);
        exit(EXIT_FAILURE);
    }

    return 0;
}



//---------------------------------------------------------------------
// Main function: setup dummy problem and run preconditioned CG.
int main(int argc, char* argv[]) {
        // Create cuSOLVER / cuBLAS handles and stream

        int deviceCount;
        hipGetDeviceCount(&deviceCount);
        std::cout << "CUDA devices available: " << deviceCount << std::endl;
        
        if (deviceCount == 0) {
            std::cerr << "No CUDA devices found! Exiting...\n";
            exit(EXIT_FAILURE);
        }
        
        // Try selecting device 0 explicitly
        int device = 0;
        hipError_t err = hipSetDevice(device);
        if (err != hipSuccess) {
            std::cerr << "Failed to set CUDA device 0: " << hipGetErrorString(err) << std::endl;
            exit(EXIT_FAILURE);
        }
        
        hipDeviceSynchronize();
        std::cout << "Using CUDA device: " << device << std::endl;
        
        
    hipsolverHandle_t cusolverH = nullptr;
    hipblasHandle_t cublasH = nullptr;
    hipStream_t stream = nullptr;


    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreate(&stream));

    //set problem params and factorization stuff using settings.json
    std::ifstream settings_file("settings.json", std::ifstream::binary);
    nlohmann::json settings = nlohmann::json::parse(settings_file);
    
    auto mat_set = settings["matrix_settings"];
    auto fact_set = settings["factorization_settings"];
    string tmp;
    tmp = mat_set["n"].dump();
    tmp = tmp.substr(1, tmp.size() - 2);
    int n = stoi(tmp);
    tmp = mat_set["condition_number"].dump();
    tmp = tmp.substr(1, tmp.size() - 2);
    double condVal = stod(tmp);
    //TODO - add code for different distributions
    tmp = fact_set["block_size"].dump();
    tmp = tmp.substr(1, tmp.size() - 2);
    int r = stoi(tmp);
    tmp = fact_set["eps_prime"].dump();
    tmp = tmp.substr(1, tmp.size() - 2);
    float eps_prime = stof(tmp);
    tmp = fact_set["floor"].dump();
    tmp = tmp.substr(1, tmp.size() - 2);
    float flr = stof(tmp);
    



    // Allocate device memory for the SPD matrix
    double* dA = nullptr;
    CUDA_CHECK(hipMalloc((void**)&dA, n*n*sizeof(double)));

    // Generate PSD with geometric distribution
    generatePSD(dA, n, condVal, DistType::Geometric, cublasH, cusolverH, stream);
    
    // (Alternatively, for arithmetic distribution, pass DistType::Arithmetic)

    // Copy back to host to inspect
    double* A = (double*) malloc(n*n*sizeof(double));
    CUDA_CHECK(hipMemcpy(A, dA, n*n*sizeof(double), hipMemcpyDeviceToHost));


    double inf_norm = 0.0;
    for (int i = 0; i < n; i++) {  // Loop over rows.
        double row_sum = 0.0;
        for (int j = 0; j < n; j++) {  // Loop over columns.
            // Since A is stored in column-major order, element (i,j) is A[i + j*n].
            row_sum += fabs(A[i + j * n]);
        }
        if (row_sum > inf_norm)
            inf_norm = row_sum;
    }
    
    // Right-hand side vector b.
    double* b = (double*) malloc(n*sizeof(double));

    for(int i = 0; i  <n; i++) b[i] = (double)rand()/(double)RAND_MAX;
    
    // Prepare containers for the solutions.
    double* x_our = (double *) malloc(n*sizeof(double));
    double* x_cusolver = (double *) malloc(n*sizeof(double));


    
    // Create CUDA events for timing.
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // ----------------------------
    // Run our custom solver.
    CUDA_CHECK(hipEventRecord(start));
    int cg_iters = precond_CG(A, x_our, b, n, r, inf_norm, eps_prime, flr);
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    float time_our = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&time_our, start, stop));
    printf("Our solver completed in %d iterations and took %.2f ms.\n", cg_iters, time_our);
    
    // ----------------------------
    // Run cuSOLVER's solver.
    CUDA_CHECK(hipEventRecord(start));
    solveWithCuSolver(A, x_cusolver, b, n);
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    float time_cusolver = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&time_cusolver, start, stop));
    printf("cuSOLVER completed in %.2f ms.\n", time_cusolver);

    // ----------------------------
    // Run MKL's solver.
    printf("Running MKL solver...\n");
    CUDA_CHECK(hipEventRecord(start));
    solveWithMKL(A, x_cusolver, b, n);
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    float time_mkl = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&time_mkl, start, stop));
    printf("MKL completed in %.2f ms.\n", time_mkl);


    // Compute residual of our solver: r_our = b - A*x_our
    double r_our_norm = 0.0;
    for (int i = 0; i < n; i++) {
        // Compute A[i,:] * x_our
        double Ai_x = 0.0;
        for (int j = 0; j < n; j++) {
            Ai_x += A[i + j * n] * x_our[j];
        }
        // residual component = b[i] - (A[i,:]*x_our)
        double ri = b[i] - Ai_x;
        r_our_norm += ri * ri;
    }
    r_our_norm = sqrt(r_our_norm);

    // Compute residual of cuSOLVER: r_cus = b - A*x_cusolver
    double r_cusolver_norm = 0.0;
    for (int i = 0; i < n; i++) {
        double Ai_x = 0.0;
        for (int j = 0; j < n; j++) {
            Ai_x += A[i + j * n] * x_cusolver[j];
        }
        double ri = b[i] - Ai_x;
        r_cusolver_norm += ri * ri;
    }
    r_cusolver_norm = sqrt(r_cusolver_norm);

    // Print out the 2-norm of both residuals
    printf("Residual norm of our solver     : %e\n", r_our_norm);
    printf("Residual norm of cuSOLVER      : %e\n", r_cusolver_norm);

    
    // ----------------------------
    // Compare the solutions (compute relative L2 norm difference).
    double diff_norm = 0.0, sol_norm = 0.0, x_max = 0.0, x_nrm = 0.0;
    for (int i = 0; i < n; i++) {
        double diff = x_our[i] - x_cusolver[i];
        diff_norm += diff * diff;
        sol_norm  += x_cusolver[i] * x_cusolver[i];
        x_nrm += x_cusolver[i]*x_cusolver[i];
        x_max = max(x_max, abs(x_cusolver[i]));
    }
    diff_norm = sqrt(diff_norm);
    sol_norm = sqrt(sol_norm);
    x_nrm = sqrt(x_nrm);
    printf("norm difference between our solver and cuSOLVER: %e\n", diff_norm/x_nrm );
    printf("inf norm of x is : %e\n", x_max);
    
    // Clean up CUDA events.
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    
    return 0;
}