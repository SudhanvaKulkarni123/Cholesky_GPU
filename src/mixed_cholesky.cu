#include "hip/hip_runtime.h"
#include "kernels_macros.cuh"
#include "matgen.cuh"
#include "micro_chol.hpp"
#include <fstream>
#include <iostream>
#include <cutlass/epilogue/thread/linear_combination_clamp.h>
#include <json.hpp>
//#include "sChol.cuh"
//#include "hChol.cuh"
//#include "fp8Chol.cuh"
#include "switching_chol.cuh"
#include <mkl.h>




int selectBestGPU() {
    int num_devices;
    hipGetDeviceCount(&num_devices);

    if (num_devices == 0) {
        std::cerr << "No CUDA devices found!" << std::endl;
        return -1;
    }

    size_t max_free_mem = 0;
    int best_device = 0;

    for (int i = 0; i < num_devices; i++) {
        hipSetDevice(i);
        size_t free_mem, total_mem;
        hipMemGetInfo(&free_mem, &total_mem);

        std::cout << "GPU " << i << ": Free = " 
                  << free_mem / (1024 * 1024) << " MB, Total = " 
                  << total_mem / (1024 * 1024) << " MB\n";

        if (free_mem > max_free_mem) {
            max_free_mem = free_mem;
            best_device = i;
        }
    }

    std::cout << "Selecting GPU " << best_device << " with max free memory: " 
              << max_free_mem / (1024 * 1024) << " MB\n";

    return best_device;
}

enum distr_type : uint8_t {
    Arith, Geom, Rand
};

void host_isnan(float* A, int n)
{
    for(int i = 0; i < n*n; i++) {
        if(isnanf(A[i])){ std::cout << "A has a nan\n"; return;}

    }
}


// vanilla CG
int vanilla_CG(double* A, double* x, double* b, int n, double normA)
{

}


// Preconditioned Conjugate Gradient (CG) solver (simplified skeleton)
// This function demonstrates the allocation, conversion, and Cholesky preconditioning.
// The actual CG loop here is simplified and uses dummy updates.
//-----------------------------
// precond_CG function
//-----------------------------
int precond_CG(double* A, double* x,
               double* b, int n, int r, double normA, float eps_prime = 0.0f, float flr = 0.0f, bool perturb_diab = false, bool left_looking = false) {


    
    std::ofstream myfile("e5m2_error_f_cond.csv");
    hipEvent_t start, stop;
        CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));


    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));  // Still creating a stream for kernels/BLAS if we like
    hipStream_t logging_stream;
    CUDA_CHECK(hipStreamCreate(&logging_stream));
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));
    CUBLAS_CHECK(hipblasSetStream(handle, stream));

    hipsolverHandle_t CuHandle_t;
    CUSOLVER_CHECK(hipsolverDnCreate(&CuHandle_t));
    CUSOLVER_CHECK(hipsolverSetStream(CuHandle_t, stream));

    


    double One = 1.0;
    float Onef = 1.0f;
    double Zero = 0.0;
    double NegOne = -1.0;

    
    hipEvent_t init_start, init_stop;
     CUDA_CHECK(hipEventCreate(&init_start));
    CUDA_CHECK(hipEventCreate(&init_stop));
    CUDA_CHECK(hipEventRecord(init_start, stream)); 
// Compute total memory required
size_t size_A_d = n * n * sizeof(double);
size_t size_A_f = n * n * sizeof(float);
size_t max_buff_size = n * r * sizeof(float);  // Matul buffer size
size_t vec_size = n * sizeof(double);
size_t half_vec_size = n * sizeof(float);  // For float vectors
size_t scal_size = sizeof(double);
size_t perm_size = n * sizeof(int);



// Total size to allocate in a single call
size_t workspace_size = n * n * sizeof(float);  // New workspace buffer

size_t total_size = 
    size_A_d +  // d_A
    size_A_f +  // s_A
    workspace_size + // New workspace buffer
    size_A_d +  // LL_copy
    7 * vec_size +  // dev_x, dev_b, dev_r, p, Ap, dev_z, dev_D (all double)
    2 * half_vec_size +  // diag_A, updated_diag (float)
    4 * scal_size +  // pAp, rz, rz_new, inf_norm_r (double)
    3 * scal_size +  // d_One, d_Zero, d_NegOne (double)
    2 * perm_size + sizeof(int);  // dev_left_perm, dev_right_perm (int)

// Allocate memory
void* d_mem = nullptr;
CUDA_CHECK(hipMalloc(&d_mem, total_size));

char* d_mem_char = reinterpret_cast<char*>(d_mem);

double* d_A        = reinterpret_cast<double*>(d_mem_char);
double* LL_copy    = reinterpret_cast<double*>(d_mem_char + size_A_d);  // LL_copy right after d_A
float* s_A         = reinterpret_cast<float*>(d_mem_char + size_A_d + size_A_d);
float* workspace   = reinterpret_cast<float*>(d_mem_char + size_A_d + size_A_d + size_A_f);  // Workspace right after s_A

double* dev_x      = reinterpret_cast<double*>(d_mem_char + size_A_d + size_A_d + size_A_f + workspace_size);
double* dev_b      = dev_x + n;
double* dev_r      = dev_b + n;
double* p          = dev_r + n;
double* Ap         = p + n;
double* dev_z      = Ap + n;
double* dev_D      = dev_z + n;

float* diag_A      = reinterpret_cast<float*>(dev_D + n);  
float* updated_diag= diag_A + n;

double* pAp        = reinterpret_cast<double*>(updated_diag + n);
double* rz         = pAp + 1;
double* rz_new     = rz + 1;
double* inf_norm_r = rz_new + 1;

double* d_One      = inf_norm_r + 1;
double* d_Zero     = d_One + 1;
double* d_NegOne   = d_Zero + 1;

int* dev_left_perm  = reinterpret_cast<int*>(d_NegOne + 1);
int* dev_right_perm = dev_left_perm + n;
int* devInfo = dev_right_perm + n;


// Copy scalar values asynchronously
CUDA_CHECK(hipMemcpyAsync(d_One,    &One,    scal_size, hipMemcpyHostToDevice, stream));
CUDA_CHECK(hipMemcpyAsync(d_Zero,   &Zero,   scal_size, hipMemcpyHostToDevice, stream));
CUDA_CHECK(hipMemcpyAsync(d_NegOne, &NegOne, scal_size, hipMemcpyHostToDevice, stream));

// Allocate host memory for preconditioner
double* D_host       = (double*)malloc(n * sizeof(double));
int* left_perm_host  = (int*)malloc(n * sizeof(int));
int* right_perm_host = (int*)malloc(n * sizeof(int));

// Initialize preconditioner data
for (int i = 0; i < n; i++) {
    left_perm_host[i] = i;
    right_perm_host[i] = i;
    D_host[i] = 1.0;  // Example
}

// Copy permutation and preconditioner data asynchronously
CUDA_CHECK(hipMemcpyAsync(dev_left_perm, left_perm_host, perm_size, hipMemcpyHostToDevice, stream));
CUDA_CHECK(hipMemcpyAsync(dev_right_perm, right_perm_host, perm_size, hipMemcpyHostToDevice, stream));
CUDA_CHECK(hipMemcpyAsync(dev_D, D_host, vec_size, hipMemcpyHostToDevice, stream));

    // Copy matrix A and vector b from host to device (synchronous).
    CUDA_CHECK(hipMemcpyAsync(d_A, A, size_A_d, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));

    CUDA_CHECK(hipMemcpyAsync(dev_b, b, vec_size, hipMemcpyHostToDevice, stream));

    // Initialize search direction p with b (synchronous).
    CUDA_CHECK(hipMemcpyAsync(p, b, vec_size, hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));

    // Convert double-precision matrix to single precision (kernel uses stream).
    int totalElems = n * n;
    int threads = 256;
    int blocks = (totalElems + threads - 1) / threads;
    convertDoubleToFloat<<<blocks, threads, 0, stream>>>(d_A, s_A, totalElems);
    CUDA_CHECK(hipStreamSynchronize(stream));


    blocks = (n + threads - 1)/threads;
    copy_diag<<<blocks, threads, 0, stream>>>(diag_A, updated_diag, s_A, n);
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipEventRecord(init_stop, stream));
    CUDA_CHECK(hipEventSynchronize(init_stop));
    float time_init = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&time_init, init_start, init_stop));
    printf("initializartion took %.2f ms.\n", time_init);
    // hipEventRecord(stop);
    // CUDA_CHECK(hipEventSynchronize(stop));
    // float time_init = 0.0f;
    // CUDA_CHECK(hipEventElapsedTime(&time_init, start, stop));
    // printf("initialization took %.2f ms.\n", time_init);



    // Perform mixed-precision Cholesky factorization on s_A (with your function).



    CUDA_CHECK(hipEventRecord(start, stream));
    CUDA_CHECK(hipDeviceSynchronize());
    //uniform_prec_GPU_cholesky(s_A, n, r, diag_A, updated_diag, handle, CuHandle_t ,stream, n);
    //uniform_prec_fused_cholesky(s_A, n, r, diag_A, updated_diag, handle, CuHandle_t ,stream, n);
    //uniform_prec_GPU_cholesky(s_A, n, r, diag_A, updated_diag, handle, CuHandle_t ,stream, n);
    //uniform_precision_Cholesky(s_A, n, r, diag_A, updated_diag, handle, stream, n);
     //halfprec_mixed_precision_Cholesky(s_A, n, r, diag_A, updated_diag, handle, CuHandle_t, stream, n, reinterpret_cast<cutlass::half_t*>(workspace));
    //fp8_mixed_precision_Cholesky(s_A, n, r, diag_A, updated_diag, handle, CuHandle_t, stream, n, reinterpret_cast<cutlass::float_e4m3_t*>(workspace), flr);

    switching_precision_Cholesky(s_A, n, r, workspace, diag_A, updated_diag, handle, CuHandle_t, stream, n, eps_prime, flr);
    CUDA_CHECK(hipStreamSynchronize(stream));
      CUDA_CHECK(hipEventRecord(stop, stream));
    CUDA_CHECK(hipEventSynchronize(stop));
    float time_our = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&time_our, start, stop));
    printf("mixed precision cholesky took %.2f ms.\n", time_our);

    //perform trmm to see if we get back same matrix

    #ifdef DEBUG_MODE
    float* debug_mat = nullptr;
    float* host_debug_mat = (float*) malloc(n*n*sizeof(float));
    CUDA_CHECK(hipMalloc((void**) &debug_mat, n*n*sizeof(float)));
    CUDA_CHECK(hipMemset(debug_mat, 0.0f, n*n*sizeof(float)));
    set_identity<<<blocks, threads, 0, stream>>>(debug_mat, n);
    CUBLAS_CHECK(hipblasStrmm(
        handle,
        HIPBLAS_SIDE_LEFT,
        HIPBLAS_FILL_MODE_LOWER,
        HIPBLAS_OP_N,
        HIPBLAS_DIAG_NON_UNIT,
        n,          // # of rows of debug_mat
        n,          // # of columns of debug_mat
        &Onef,
        s_A, n,     // pointer to L
        debug_mat, n,
        debug_mat, n
    ));

    CUDA_CHECK(hipStreamSynchronize(stream));
    CUBLAS_CHECK(hipblasStrmm(
        handle,
        HIPBLAS_SIDE_LEFT,
        HIPBLAS_FILL_MODE_LOWER,
        HIPBLAS_OP_T,      // now use the transpose of L
        HIPBLAS_DIAG_NON_UNIT,
        n,  // # rows
        n,  // # cols
        &Onef,
        s_A, n,
        debug_mat, n,
        debug_mat, n
    ));
    CUDA_CHECK(hipStreamSynchronize(stream));



    #endif

    CUDA_CHECK(hipStreamSynchronize(stream));
    

    CUBLAS_CHECK(hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE));


    // Convert factorized single-precision matrix back to double
    convertFloattoDouble<<<blocks, threads, 0, stream>>>(s_A, LL_copy, totalElems);
    CUDA_CHECK(hipStreamSynchronize(stream));

    // Solve L y = b then L^T x = y; use p as temp
    float time_init_soln;
    hipEventRecord(start, stream);
    CUSOLVER_CHECK(hipsolverDnDpotrs(CuHandle_t, HIPBLAS_FILL_MODE_LOWER, n, 1, LL_copy, n, dev_z, n, devInfo));

    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_init_soln, start, stop);


    // Set dev_x = p (device to device copy - synchronous).
    CUDA_CHECK(hipMemcpy(dev_x, p, vec_size, hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipStreamSynchronize(stream));

    // r = b - A*x
    CUBLAS_CHECK(hipblasDgemv(handle, HIPBLAS_OP_N, n, n, d_NegOne,
                             d_A, n, dev_x, 1, d_One, dev_b, 1));           //doesnt matter if its row or col major
    CUDA_CHECK(hipMemcpy(dev_r, dev_b, vec_size, hipMemcpyDeviceToDevice));
    CUDA_CHECK(hipStreamSynchronize(stream));

     double cpu_inf_norm_r = 0.0;

    int vecThreads = 256;
    int vecBlocks = (n + vecThreads - 1) / vecThreads;

        // Allocate temporary device memory to store per-block maximums.
    double* dev_blockMax = nullptr;
    CUDA_CHECK(hipMalloc((void**)&dev_blockMax, vecBlocks * sizeof(double)));

    vanilla_Max<<<vecBlocks, vecThreads, vecThreads * sizeof(double), stream>>>(dev_r, dev_blockMax, n);

    int finalThreads = 256;

    final_max_reduce<<<1, finalThreads, finalThreads * sizeof(double), stream>>>(dev_blockMax, inf_norm_r, vecBlocks);

    // Wait for the kernels to finish.
    CUDA_CHECK(hipStreamSynchronize(stream));

    // Free the temporary block maximum array.
    CUDA_CHECK(hipFree(dev_blockMax));

    CUDA_CHECK(hipStreamSynchronize(stream));

       
    // synchronous memcpy from device to host
    CUDA_CHECK(hipMemcpy(&cpu_inf_norm_r, inf_norm_r, sizeof(double), hipMemcpyDeviceToHost));

    double init_residual = cpu_inf_norm_r / normA;
    std::cout << "init residual is : " << init_residual << std::endl;


    float time_init_precond;
    // Compute r^T z
    hipEventRecord(start, stream);
    CUDA_CHECK(hipMemcpy(dev_z, dev_r, vec_size, hipMemcpyDeviceToDevice));
    CUSOLVER_CHECK(hipsolverDnDpotrs(CuHandle_t, HIPBLAS_FILL_MODE_LOWER, n, 1, LL_copy, n, dev_z, n, devInfo));


    CUBLAS_CHECK(hipblasDdot(handle, n, dev_z, 1, dev_r, 1, rz));

    CUDA_CHECK(hipMemcpy(p, dev_z, vec_size, hipMemcpyDeviceToDevice));
    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_init_precond, start, stop);

    const double d_conv_bound = 1e-14;
    int count = 0;

    float gemv_CG_time = 0;
    float trsv_CG_time = 0;

    float update_search_time = 0;

    // CG loop
    const int max_CG_iter = 1000;
    for (int j = 0; j < max_CG_iter; j++) {
        // Ap = A * p
        hipEventRecord(start, stream);
        CUBLAS_CHECK(hipblasDgemv(handle, HIPBLAS_OP_N, n, n,
                                 d_One, d_A, n, p, 1, d_Zero, Ap, 1));          //doesnt matter if its row or col major
        
        CUBLAS_CHECK(hipblasDdot(handle, n, Ap, 1, p, 1, pAp));
        CUDA_CHECK(hipStreamSynchronize(stream));
        hipEventRecord(stop, stream);
        hipEventSynchronize(stop);
        float elapsed_time;
        hipEventElapsedTime(&elapsed_time, start, stop);
        gemv_CG_time += elapsed_time;



        // x_r_update kernel
  
        x_r_update<<<vecBlocks, vecThreads, sizeof(double), stream>>>(dev_x, dev_r, rz, pAp, p, Ap, n);
        CUDA_CHECK(hipStreamSynchronize(stream));

        // Compute infinity norm of residual
         vanilla_Max<<<vecBlocks, vecThreads, vecThreads * sizeof(double), stream>>>(dev_r, dev_blockMax, n);
         final_max_reduce<<<1, finalThreads, finalThreads * sizeof(double), stream>>>(dev_blockMax, inf_norm_r, vecBlocks);
        CUDA_CHECK(hipStreamSynchronize(stream));

       
        // synchronous memcpy from device to host
        CUDA_CHECK(hipMemcpy(&cpu_inf_norm_r, inf_norm_r, sizeof(double), hipMemcpyDeviceToHost));

        double scaled_residual = cpu_inf_norm_r / normA;
        myfile << j << "," << scaled_residual << std::endl;
        if (scaled_residual < d_conv_bound) {
            count = j + 1;
            std::cout << "Converged in " << count << " iterations." << std::endl;

            break;
        }

        // Preconditioning
 
        CUDA_CHECK(hipMemcpy(dev_z, dev_r, vec_size, hipMemcpyDeviceToDevice));
        CUDA_CHECK(hipStreamSynchronize(stream));

        // Scale dev_z by D
        //diag_scal<<<vecBlocks, vecThreads, 0, stream>>>(dev_z, dev_z, dev_D, n);
        CUDA_CHECK(hipStreamSynchronize(stream));

        hipEventRecord(start, stream);
        CUSOLVER_CHECK(hipsolverDnDpotrs(CuHandle_t, HIPBLAS_FILL_MODE_LOWER, n, 1, LL_copy, n, dev_z, n, devInfo));
        hipEventRecord(stop, stream);
        hipEventSynchronize(stop);
        float elapsed_time2;
        hipEventElapsedTime(&elapsed_time2, start, stop);
        trsv_CG_time += elapsed_time2;

        // Scale dev_z by D again
        // diag_scal<<<vecBlocks, vecThreads, 0, stream>>>(dev_z, dev_z, dev_D, n);
        CUDA_CHECK(hipStreamSynchronize(stream));

        // Compute r^T z
        CUBLAS_CHECK(hipblasDdot(handle, n, dev_r, 1, dev_z, 1, rz_new));

        // p = z + beta * p
        hipEventRecord(start, stream);
        update_search_dir<<<vecBlocks, vecThreads, sizeof(double), stream>>>(p, dev_z, rz_new, rz, n);
        CUDA_CHECK(hipStreamSynchronize(stream));
        hipEventRecord(stop, stream);
        hipEventSynchronize(stop);
        float elapsed_time3;
        hipEventElapsedTime(&elapsed_time3, start, stop);
        update_search_time += elapsed_time3;

        CUDA_CHECK(hipMemcpy(rz, rz_new, sizeof(double), hipMemcpyDeviceToDevice));
        CUDA_CHECK(hipStreamSynchronize(stream));
        count++;
    }


    printf("GEMVs in CG took %.2f ms.\n", gemv_CG_time);
    printf("trsv in CG took %.2f ms.\n", trsv_CG_time);
    printf("search dir update in CG took %.2f ms.\n", update_search_time);




    float epilogue_time;
    // Copy final solution to host
    CUDA_CHECK(hipMemcpy(x, dev_x, vec_size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipDeviceSynchronize());

    // Clean up
   CUDA_CHECK(hipFree(d_mem));

    CUBLAS_CHECK(hipblasDestroy(handle));
    CUDA_CHECK(hipStreamDestroy(stream));
    CUDA_CHECK(hipStreamDestroy(logging_stream));



    myfile.close();

    return count;
    } // End CG loop.


    int vanilla_CG(double* d_A, double* d_x, double* d_b, int n, double tol = 1e-6, int max_iter = 1000) {
        hipblasHandle_t handle;
        hipblasCreate(&handle);
    
        // Allocate GPU memory
        double *d_r, *d_p, *d_Ap;
        hipMalloc((void**)&d_r, n * sizeof(double));
        hipMalloc((void**)&d_p, n * sizeof(double));
        hipMalloc((void**)&d_Ap, n * sizeof(double));
    
        double alpha, beta, r_dot, r_dot_new, pAp;
        double neg_one = -1.0, one = 1.0, zero = 0.0;
    
        // Initialize r = b - Ax
        hipblasDgemv(handle, HIPBLAS_OP_N, n, n, &one, d_A, n, d_x, 1, &zero, d_r, 1);
        hipblasDaxpy(handle, n, &neg_one, d_r, 1, d_b, 1);  // r = b - Ax
        hipMemcpy(d_p, d_r, n * sizeof(double), hipMemcpyDeviceToDevice); // p = r
    
        // Compute initial residual norm r_dot = r^T r
        hipblasDdot(handle, n, d_r, 1, d_r, 1, &r_dot);
        double b_norm;
        hipblasDdot(handle, n, d_b, 1, d_b, 1, &b_norm);
        b_norm = sqrt(b_norm);
    
        int k = 0;
        while (sqrt(r_dot) / b_norm > tol && k < max_iter) {
            // Ap = A * p
            hipblasDgemv(handle, HIPBLAS_OP_N, n, n, &one, d_A, n, d_p, 1, &zero, d_Ap, 1);
            
            // Compute alpha = r^T r / (p^T A p)
            hipblasDdot(handle, n, d_p, 1, d_Ap, 1, &pAp);
            alpha = r_dot / pAp;
    
            // x = x + alpha * p
            hipblasDaxpy(handle, n, &alpha, d_p, 1, d_x, 1);
    
            // r = r - alpha * Ap
            double negalpha = -alpha;
            hipblasDaxpy(handle, n, &(negalpha), d_Ap, 1, d_r, 1);
    
            // Compute new r_dot = r^T r
            hipblasDdot(handle, n, d_r, 1, d_r, 1, &r_dot_new);
    
            // Compute beta = (new r^T r) / (old r^T r)
            beta = r_dot_new / r_dot;
    
            // p = r + beta * p
            hipblasDscal(handle, n, &beta, d_p, 1);
            hipblasDaxpy(handle, n, &one, d_r, 1, d_p, 1);
    
            // Update r_dot for next iteration
            r_dot = r_dot_new;
            k++;
        }
    
        // Cleanup
        hipFree(d_r);
        hipFree(d_p);
        hipFree(d_Ap);
        hipblasDestroy(handle);
    
        return k; // Return number of iterations
    }

// Solve A*x = b using cuSOLVER's Cholesky factorization.
int solveWithCuSolver(double* A, double* x, const double* b, int n) {
    hipsolverHandle_t cusolverH = nullptr;
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
    
    double *d_A = nullptr;
    double *d_b = nullptr;
    int lda = n;
    size_t matrixSize = n * n * sizeof(double);
    size_t vecSize = n * sizeof(double);
    CUDA_CHECK(hipMalloc((void**)&d_A, matrixSize));
    CUDA_CHECK(hipMalloc((void**)&d_b, vecSize));
    
    // Copy A and b from host to device.
    CUDA_CHECK(hipMemcpy(d_A, A, matrixSize, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, b, vecSize, hipMemcpyHostToDevice));
    
    int work_size = 0;
    int *devInfo = nullptr;
    CUDA_CHECK(hipMalloc((void**)&devInfo, sizeof(int)));
    
    // Query working space for Cholesky factorization.
    hipsolverDnDpotrf_bufferSize(cusolverH, HIPBLAS_FILL_MODE_LOWER, n, d_A, lda, &work_size);
    double *work = nullptr;
    CUDA_CHECK(hipMalloc((void**)&work, work_size * sizeof(double)));
    
    // Cholesky factorization: A = L * L^T.
    hipsolverDnDpotrf(cusolverH, HIPBLAS_FILL_MODE_LOWER, n, d_A, lda, work, work_size, devInfo);
    int devInfo_h = 0;
    CUDA_CHECK(hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    if (devInfo_h != 0) {
        printf("Cholesky factorization failed with devInfo = %d\n", devInfo_h);
        exit(EXIT_FAILURE);
    }
    
    // Solve A*x = b.
    hipsolverDnDpotrs(cusolverH, HIPBLAS_FILL_MODE_LOWER, n, 1, d_A, lda, d_b, n, devInfo);
    CUDA_CHECK(hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    if (devInfo_h != 0) {
        printf("Cholesky solve failed with devInfo = %d\n", devInfo_h);
        exit(EXIT_FAILURE);
    }
    
    // Copy the solution from device to host.
    CUDA_CHECK(hipMemcpy(x, d_b, vecSize, hipMemcpyDeviceToHost));
    
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(work));
    CUDA_CHECK(hipFree(devInfo));
    hipsolverDnDestroy(cusolverH);
    
    return 0;
}


// Solve A*x = b using Intel MKL's Cholesky factorization.
int solveWithMKL(double* A, double* x, const double* b, int n) {
    // Copy b to x since LAPACK overwrites b with the solution.
    memcpy(x, b, n * sizeof(double));

    // Perform Cholesky factorization (A = L * L^T)
    int info = LAPACKE_dpotrf(LAPACK_COL_MAJOR, 'L', n, A, n);
    if (info != 0) {
        printf("MKL Cholesky factorization failed with info = %d\n", info);
        exit(EXIT_FAILURE);
    }

    // Solve A*x = b using the factorized matrix
    info = LAPACKE_dpotrs(LAPACK_COL_MAJOR, 'L', n, 1, A, n, x, n);
    if (info != 0) {
        printf("MKL Cholesky solve failed with info = %d\n", info);
        exit(EXIT_FAILURE);
    }

    return 0;
}



//---------------------------------------------------------------------
// Main function: setup dummy problem and run preconditioned CG.
int main(int argc, char* argv[]) {
        // Create cuSOLVER / cuBLAS handles and stream

        int deviceCount;
        hipGetDeviceCount(&deviceCount);
        std::cout << "CUDA devices available: " << deviceCount << std::endl;
        
        if (deviceCount == 0) {
            std::cerr << "No CUDA devices found! Exiting...\n";
            exit(EXIT_FAILURE);
        }
        
        int num_devices;
hipGetDeviceCount(&num_devices);


        // Try selecting device 1 explicitly
        int best_gpu = selectBestGPU();
        if (best_gpu >= 0) {
            hipSetDevice(best_gpu);  // Set the selected GPU
            std::cout << "Using GPU " << best_gpu << std::endl;
        }
        hipDeviceSynchronize();

        
        
    hipsolverHandle_t cusolverH = nullptr;
    hipblasHandle_t cublasH = nullptr;
    hipStream_t stream = nullptr;


    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
    CUBLAS_CHECK(hipblasCreate(&cublasH));
    CUDA_CHECK(hipStreamCreate(&stream));

    //set problem params and factorization stuff using settings.json
    std::ifstream settings_file("settings.json", std::ifstream::binary);
    nlohmann::json settings = nlohmann::json::parse(settings_file);

   
    
    auto mat_set = settings["matrix_settings"];
    auto fact_set = settings["factorization_settings"];
    string tmp;
    tmp = mat_set["n"].dump();
    tmp = tmp.substr(1, tmp.size() - 2);
    int n = stoi(tmp);
    tmp = mat_set["condition_number"].dump();
    tmp = tmp.substr(1, tmp.size() - 2);
    double condVal = stod(tmp);
    //TODO - add code for different distributions
    tmp = fact_set["block_size"].dump();
    tmp = tmp.substr(1, tmp.size() - 2);
    int r = stoi(tmp);
    tmp = fact_set["eps_prime"].dump();
    tmp = tmp.substr(1, tmp.size() - 2);
    float eps_prime = stof(tmp);
    tmp = fact_set["floor"].dump();
    tmp = tmp.substr(1, tmp.size() - 2);
    float flr = stof(tmp);
    tmp = fact_set["diag_pert"].dump();
    tmp = tmp.substr(1, tmp.size() - 2);
    bool perturb_diag = stoi(tmp) != 0;
    tmp = fact_set["left"].dump();
    tmp = tmp.substr(1, tmp.size() - 2);
    bool left_looking = stoi(tmp) != 0;

    



    // Allocate device memory for the SPD matrix
    double* dA = nullptr;
    CUDA_CHECK(hipMalloc((void**)&dA, n*n*sizeof(double)));

    // Generate PSD with geometric distribution
    generatePSD(dA, n, condVal, DistType::Geometric, cublasH, cusolverH, stream);
    
    // (Alternatively, for arithmetic distribution, pass DistType::Arithmetic)

    // Copy back to host to inspect
    double* A = (double*) malloc(n*n*sizeof(double));
    CUDA_CHECK(hipMemcpy(A, dA, n*n*sizeof(double), hipMemcpyDeviceToHost));


    double inf_norm = 0.0;
    for (int i = 0; i < n; i++) {  // Loop over rows.
        double row_sum = 0.0;
        for (int j = 0; j < n; j++) {  // Loop over columns.
            // Since A is stored in column-major order, element (i,j) is A[i + j*n].
            row_sum += fabs(A[i + j * n]);
        }
        if (row_sum > inf_norm)
            inf_norm = row_sum;
    }
    
    // Right-hand side vector b.
    double* b = (double*) malloc(n*sizeof(double));

    for(int i = 0; i  <n; i++) b[i] = (double)rand()/(double)RAND_MAX;
    
    // Prepare containers for the solutions.
    double* x_our = (double *) malloc(n*sizeof(double));
    double* x_cusolver = (double *) malloc(n*sizeof(double));


    
    // Create CUDA events for timing.
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // ----------------------------
    // Run our custom solver.
    CUDA_CHECK(hipEventRecord(start));
    int cg_iters = precond_CG(A, x_our, b, n, r, inf_norm, eps_prime, flr, perturb_diag, left_looking);
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    float time_our = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&time_our, start, stop));
    printf("Our solver completed in %d iterations and took %.2f ms.\n", cg_iters, time_our);
    
    // ----------------------------
    // Run cuSOLVER's solver.
    CUDA_CHECK(hipEventRecord(start));
    solveWithCuSolver(A, x_cusolver, b, n);
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    float time_cusolver = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&time_cusolver, start, stop));
    printf("cuSOLVER completed in %.2f ms.\n", time_cusolver);

    // ----------------------------
    // // Run MKL's solver.
    // printf("Running MKL solver...\n");
    // CUDA_CHECK(hipEventRecord(start));
    // solveWithMKL(A, x_cusolver, b, n);
    // CUDA_CHECK(hipEventRecord(stop));
    // CUDA_CHECK(hipEventSynchronize(stop));
    // float time_mkl = 0.0f;
    // CUDA_CHECK(hipEventElapsedTime(&time_mkl, start, stop));
    // printf("MKL completed in %.2f ms.\n", time_mkl);


    // Compute residual of our solver: r_our = b - A*x_our
    double r_our_norm = 0.0;
    for (int i = 0; i < n; i++) {
        // Compute A[i,:] * x_our
        double Ai_x = 0.0;
        for (int j = 0; j < n; j++) {
            Ai_x += A[i + j * n] * x_our[j];
        }
        // residual component = b[i] - (A[i,:]*x_our)
        double ri = b[i] - Ai_x;
        r_our_norm += ri * ri;
    }
    r_our_norm = sqrt(r_our_norm);

    // Compute residual of cuSOLVER: r_cus = b - A*x_cusolver
    double r_cusolver_norm = 0.0;
    for (int i = 0; i < n; i++) {
        double Ai_x = 0.0;
        for (int j = 0; j < n; j++) {
            Ai_x += A[i + j * n] * x_cusolver[j];
        }
        double ri = b[i] - Ai_x;
        r_cusolver_norm += ri * ri;
    }
    r_cusolver_norm = sqrt(r_cusolver_norm);

    // Print out the 2-norm of both residuals
    printf("Residual norm of our solver     : %e\n", r_our_norm);
    printf("Residual norm of cuSOLVER      : %e\n", r_cusolver_norm);

    CUDA_CHECK(hipEventRecord(start));

    double *d_x_vanilla, *d_b;
    hipMalloc((void**)&d_x_vanilla, n * sizeof(double));
    hipMalloc((void**)&d_b, n * sizeof(double));
    hipMemcpy(d_b, b, n * sizeof(double), hipMemcpyHostToDevice);

    double* x_vanilla = (double*)malloc(n * sizeof(double)); // Fixed sizeof typo

    // Run Vanilla CG
    CUDA_CHECK(hipEventRecord(start));
    int vanilla_cg_iters = vanilla_CG(dA, d_x_vanilla, d_b, n);
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));

    float time_vanilla_cg = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&time_vanilla_cg, start, stop));
    printf("Vanilla CG completed in %d iterations and took %.2f ms.\n", vanilla_cg_iters, time_vanilla_cg);

    // ----------------------------
    // Compute residual norm for Vanilla CG
    double r_vanilla_cg_norm = 0.0;
    for (int i = 0; i < n; i++) {
        double Ai_x = 0.0;
        for (int j = 0; j < n; j++) {
            Ai_x += A[i + j * n] * x_vanilla[j];
        }
        double ri = b[i] - Ai_x;
        r_vanilla_cg_norm += ri * ri;
    }
    r_vanilla_cg_norm = sqrt(r_vanilla_cg_norm);

    // Print out the residual norm for Vanilla CG
    printf("Residual norm of Vanilla CG    : %e\n", r_vanilla_cg_norm);


    
    // ----------------------------
    // Compare the solutions (compute relative L2 norm difference).
    double diff_norm = 0.0, sol_norm = 0.0, x_max = 0.0, x_nrm = 0.0;
    for (int i = 0; i < n; i++) {
        double diff = x_our[i] - x_cusolver[i];
        diff_norm += diff * diff;
        sol_norm  += x_cusolver[i] * x_cusolver[i];
        x_nrm += x_cusolver[i]*x_cusolver[i];
        x_max = max(x_max, abs(x_cusolver[i]));
    }
    diff_norm = sqrt(diff_norm);
    sol_norm = sqrt(sol_norm);
    x_nrm = sqrt(x_nrm);
    printf("norm difference between our solver and cuSOLVER: %e\n", diff_norm/x_nrm );
    printf("inf norm of x is : %e\n", x_max);
    
    // Clean up CUDA events.
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    hipFree(d_x_vanilla);
    hipFree(d_b);
    free(x_vanilla);

    
    return 0;
}